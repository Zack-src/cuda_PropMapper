﻿
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <iomanip>

int main()
{
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        std::cerr << "Aucun GPU détecté." << std::endl;
        return -1;
    }

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    for (int device = 0; device < deviceCount; device++) {
    
        std::cout << "name " << prop.name << std::endl;
    
        std::cout << "uuid ";
        for (int i = 0; i < 16; i++) {
            std::cout << std::hex << std::setw(2) << std::setfill('0')
                << static_cast<unsigned int>(static_cast<unsigned char>(prop.uuid.bytes[i]));
        }
        std::cout << std::dec << std::endl;

        std::cout << "luid " << prop.luid << std::endl;
        std::cout << "luidDeviceNodeMask " << prop.luidDeviceNodeMask << std::endl;
        std::cout << "totalGlobalMem " << prop.totalGlobalMem << std::endl;
        std::cout << "sharedMemPerBlock " << prop.sharedMemPerBlock << std::endl;
        std::cout << "regsPerBlock " << prop.regsPerBlock << std::endl;
        std::cout << "warpSize " << prop.warpSize << std::endl;
        std::cout << "memPitch " << prop.memPitch << std::endl;
        std::cout << "maxThreadsPerBlock " << prop.maxThreadsPerBlock << std::endl;

        std::cout << "maxThreadsDimX " << prop.maxThreadsDim[0] << std::endl;
        std::cout << "maxThreadsDimY " << prop.maxThreadsDim[1] << std::endl;
        std::cout << "maxThreadsDimZ " << prop.maxThreadsDim[2] << std::endl;

        std::cout << "maxGridSizeX " << prop.maxGridSize[0] << std::endl;
        std::cout << "maxGridSizeY " << prop.maxGridSize[1] << std::endl;
        std::cout << "maxGridSizeZ " << prop.maxGridSize[2] << std::endl;

        std::cout << "clockRate " << prop.clockRate << std::endl;
        std::cout << "totalConstMem " << prop.totalConstMem << std::endl;
        std::cout << "major " << prop.major << std::endl;
        std::cout << "minor " << prop.minor << std::endl;
        std::cout << "textureAlignment " << prop.textureAlignment << std::endl;
        std::cout << "texturePitchAlignment " << prop.texturePitchAlignment << std::endl;
        std::cout << "deviceOverlap " << prop.deviceOverlap << std::endl;
        std::cout << "multiProcessorCount " << prop.multiProcessorCount << std::endl;
        std::cout << "kernelExecTimeoutEnabled " << prop.kernelExecTimeoutEnabled << std::endl;
        std::cout << "integrated " << prop.integrated << std::endl;
        std::cout << "canMapHostMemory " << prop.canMapHostMemory << std::endl;
        std::cout << "computeMode " << prop.computeMode << std::endl;
        std::cout << "maxTexture1D " << prop.maxTexture1D << std::endl;
        std::cout << "maxTexture1DMipmap " << prop.maxTexture1DMipmap << std::endl;
        std::cout << "maxTexture1DLinear " << prop.maxTexture1DLinear << std::endl;
    
        std::cout << "maxTexture2D.W " << prop.maxTexture2D[0] << std::endl;
        std::cout << "maxTexture2D.H " << prop.maxTexture2D[1] << std::endl;
    
        std::cout << "maxTexture2DMipmap.W " << prop.maxTexture2DMipmap[0] << std::endl;
        std::cout << "maxTexture2DMipmap.H " << prop.maxTexture2DMipmap[1] << std::endl;
    
        std::cout << "maxTexture2DLinear.W " << prop.maxTexture2DLinear[0] << std::endl;
        std::cout << "maxTexture2DLinear.H " << prop.maxTexture2DLinear[1] << std::endl;
        std::cout << "maxTexture2DLinear.L " << prop.maxTexture2DLinear[2] << std::endl;

        std::cout << "maxTexture2DGather.W " << prop.maxTexture2DGather[0] << std::endl;
        std::cout << "maxTexture2DGather.H " << prop.maxTexture2DGather[1] << std::endl;

        std::cout << "maxTexture3D.W " << prop.maxTexture3D[0] << std::endl;
        std::cout << "maxTexture3D.H " << prop.maxTexture3D[1] << std::endl;
        std::cout << "maxTexture3D.L " << prop.maxTexture3D[2] << std::endl;

        std::cout << "maxTexture3DAlt.W " << prop.maxTexture3DAlt[0] << std::endl;
        std::cout << "maxTexture3DAlt.H " << prop.maxTexture3DAlt[1] << std::endl;
        std::cout << "maxTexture3DAlt.L " << prop.maxTexture3DAlt[2] << std::endl;

        std::cout << "maxTextureCubemap " << prop.maxTextureCubemap << std::endl;

        std::cout << "maxTexture1DLayered.W " << prop.maxTexture1DLayered[0] << std::endl;
        std::cout << "maxTexture1DLayered.H " << prop.maxTexture1DLayered[1] << std::endl;

        std::cout << "maxTexture2DLayered.W " << prop.maxTexture2DLayered[0] << std::endl;
        std::cout << "maxTexture2DLayered.H " << prop.maxTexture2DLayered[1] << std::endl;
        std::cout << "maxTexture2DLayered.L " << prop.maxTexture2DLayered[2] << std::endl;

        std::cout << "maxTextureCubemapLayered.W " << prop.maxTextureCubemapLayered[0] << std::endl;
        std::cout << "maxTextureCubemapLayered.H " << prop.maxTextureCubemapLayered[1] << std::endl;
    
        std::cout << "maxSurface1D " << prop.maxSurface1D << std::endl;
    
        std::cout << "maxSurface2D.W " << prop.maxSurface2D[0] << std::endl;
        std::cout << "maxSurface2D.H " << prop.maxSurface2D[1] << std::endl;
    
        std::cout << "maxSurface3D.W " << prop.maxSurface3D[0] << std::endl;
        std::cout << "maxSurface3D.H " << prop.maxSurface3D[1] << std::endl;
        std::cout << "maxSurface3D.L " << prop.maxSurface3D[2] << std::endl;
    
        std::cout << "maxSurface1DLayered.W " << prop.maxSurface1DLayered[0] << std::endl;
        std::cout << "maxSurface1DLayered.H " << prop.maxSurface1DLayered[1] << std::endl;
    
        std::cout << "maxSurface2DLayered.W " << prop.maxSurface2DLayered[0] << std::endl;
        std::cout << "maxSurface2DLayered.H " << prop.maxSurface2DLayered[1] << std::endl;
        std::cout << "maxSurface2DLayered.L " << prop.maxSurface2DLayered[2] << std::endl;
    
        std::cout << "maxSurfaceCubemap " << prop.maxSurfaceCubemap << std::endl;

        std::cout << "maxSurfaceCubemapLayered.W " << prop.maxSurfaceCubemapLayered[0] << std::endl;
        std::cout << "maxSurfaceCubemapLayered.H " << prop.maxSurfaceCubemapLayered[1] << std::endl;
    
        std::cout << "surfaceAlignment " << prop.surfaceAlignment << std::endl;
        std::cout << "concurrentKernels " << prop.concurrentKernels << std::endl;
        std::cout << "ECCEnabled " << prop.ECCEnabled << std::endl;
        std::cout << "pciBusID " << prop.pciBusID << std::endl;
        std::cout << "pciDeviceID " << prop.pciDeviceID << std::endl;
        std::cout << "pciDomainID " << prop.pciDomainID << std::endl;
        std::cout << "tccDriver " << prop.tccDriver << std::endl;
        std::cout << "asyncEngineCount " << prop.asyncEngineCount << std::endl;
        std::cout << "unifiedAddressing " << prop.unifiedAddressing << std::endl;
        std::cout << "memoryClockRate " << prop.memoryClockRate << std::endl;
        std::cout << "memoryBusWidth " << prop.memoryBusWidth << std::endl;
        std::cout << "l2CacheSize " << prop.l2CacheSize << std::endl;
        std::cout << "persistingL2CacheMaxSize " << prop.persistingL2CacheMaxSize << std::endl;
        std::cout << "maxThreadsPerMultiProcessor " << prop.maxThreadsPerMultiProcessor << std::endl;
        std::cout << "streamPrioritiesSupported " << prop.streamPrioritiesSupported << std::endl;
        std::cout << "globalL1CacheSupported " << prop.globalL1CacheSupported << std::endl;
        std::cout << "localL1CacheSupported " << prop.localL1CacheSupported << std::endl;
        std::cout << "sharedMemPerMultiprocessor " << prop.sharedMemPerMultiprocessor << std::endl;
        std::cout << "regsPerMultiprocessor " << prop.regsPerMultiprocessor << std::endl;
        std::cout << "managedMemory " << prop.managedMemory << std::endl;
        std::cout << "isMultiGpuBoard " << prop.isMultiGpuBoard << std::endl;
        std::cout << "multiGpuBoardGroupID " << prop.multiGpuBoardGroupID << std::endl;
        std::cout << "hostNativeAtomicSupported " << prop.hostNativeAtomicSupported << std::endl;
        std::cout << "singleToDoublePrecisionPerfRatio " << prop.singleToDoublePrecisionPerfRatio << std::endl;
        std::cout << "pageableMemoryAccess " << prop.pageableMemoryAccess << std::endl;
        std::cout << "concurrentManagedAccess " << prop.concurrentManagedAccess << std::endl;
        std::cout << "computePreemptionSupported " << prop.computePreemptionSupported << std::endl;
        std::cout << "canUseHostPointerForRegisteredMem " << prop.canUseHostPointerForRegisteredMem << std::endl;
        std::cout << "cooperativeLaunch " << prop.cooperativeLaunch << std::endl;
        std::cout << "cooperativeMultiDeviceLaunch " << prop.cooperativeMultiDeviceLaunch << std::endl;
        std::cout << "sharedMemPerBlockOptin " << prop.sharedMemPerBlockOptin << std::endl;
        std::cout << "pageableMemoryAccessUsesHostPageTa " << prop.pageableMemoryAccessUsesHostPageTables << std::endl;
        std::cout << "directManagedMemAccessFromHost " << prop.directManagedMemAccessFromHost << std::endl;
        std::cout << "maxBlocksPerMultiProcessor " << prop.maxBlocksPerMultiProcessor << std::endl;
        std::cout << "accessPolicyMaxWindowSize " << prop.accessPolicyMaxWindowSize << std::endl;
        std::cout << "reservedSharedMemPerBlock " << prop.reservedSharedMemPerBlock << std::endl;
        std::cout << "hostRegisterSupported " << prop.hostRegisterSupported << std::endl;
        std::cout << "sparseCudaArraySupported " << prop.sparseCudaArraySupported << std::endl;
        std::cout << "hostRegisterReadOnlySupported " << prop.hostRegisterReadOnlySupported << std::endl;
        std::cout << "timelineSemaphoreInteropSupported " << prop.timelineSemaphoreInteropSupported << std::endl;
        std::cout << "memoryPoolsSupported " << prop.memoryPoolsSupported << std::endl;
        std::cout << "gpuDirectRDMASupported " << prop.gpuDirectRDMASupported << std::endl;
        std::cout << "gpuDirectRDMAFlushWritesOptions " << prop.gpuDirectRDMAFlushWritesOptions << std::endl;
        std::cout << "gpuDirectRDMAWritesOrdering " << prop.gpuDirectRDMAWritesOrdering << std::endl;
        std::cout << "memoryPoolSupportedHandleTypes " << prop.memoryPoolSupportedHandleTypes << std::endl;
        std::cout << "deferredMappingCudaArraySupported " << prop.deferredMappingCudaArraySupported << std::endl;
        std::cout << "ipcEventSupported " << prop.ipcEventSupported << std::endl;
        std::cout << "clusterLaunch " << prop.clusterLaunch << std::endl;
        std::cout << "unifiedFunctionPointers " << prop.unifiedFunctionPointers << std::endl;

        std::cout << "reserved2.W " << prop.reserved2[0] << std::endl;
        std::cout << "reserved2.H " << prop.reserved2[1] << std::endl;

        std::cout << "reserved1 " << prop.reserved1[1] << std::endl;

        std::cout << "reserved.0 " << prop.reserved[0] << std::endl;
        std::cout << "reserved.1 " << prop.reserved[1] << std::endl;
        std::cout << "reserved.2 " << prop.reserved[2] << std::endl;
        std::cout << "reserved.3 " << prop.reserved[3] << std::endl;
        std::cout << "reserved.4 " << prop.reserved[4] << std::endl;
        std::cout << "reserved.5 " << prop.reserved[5] << std::endl;
        std::cout << "reserved.6 " << prop.reserved[6] << std::endl;
        std::cout << "reserved.7 " << prop.reserved[7] << std::endl;
        std::cout << "reserved.8 " << prop.reserved[8] << std::endl;
        std::cout << "reserved.9 " << prop.reserved[9] << std::endl;
        std::cout << "reserved.10 " << prop.reserved[10] << std::endl;
        std::cout << "reserved.11 " << prop.reserved[11] << std::endl;
        std::cout << "reserved.12 " << prop.reserved[12] << std::endl;
        std::cout << "reserved.13 " << prop.reserved[13] << std::endl;
        std::cout << "reserved.14 " << prop.reserved[14] << std::endl;
        std::cout << "reserved.15 " << prop.reserved[15] << std::endl;
        std::cout << "reserved.16 " << prop.reserved[16] << std::endl;
        std::cout << "reserved.17 " << prop.reserved[17] << std::endl;
        std::cout << "reserved.18 " << prop.reserved[18] << std::endl;
        std::cout << "reserved.19 " << prop.reserved[19] << std::endl;
        std::cout << "reserved.20 " << prop.reserved[20] << std::endl;
        std::cout << "reserved.21 " << prop.reserved[21] << std::endl;
        std::cout << "reserved.22 " << prop.reserved[22] << std::endl;
        std::cout << "reserved.23 " << prop.reserved[23] << std::endl;
        std::cout << "reserved.24 " << prop.reserved[24] << std::endl;
        std::cout << "reserved.25 " << prop.reserved[25] << std::endl;
        std::cout << "reserved.26 " << prop.reserved[26] << std::endl;
        std::cout << "reserved.27 " << prop.reserved[27] << std::endl;
        std::cout << "reserved.28 " << prop.reserved[28] << std::endl;
        std::cout << "reserved.29 " << prop.reserved[29] << std::endl;
        std::cout << "reserved.30 " << prop.reserved[30] << std::endl;
        std::cout << "reserved.31 " << prop.reserved[31] << std::endl;
        std::cout << "reserved.32 " << prop.reserved[32] << std::endl;
        std::cout << "reserved.33 " << prop.reserved[33] << std::endl;
        std::cout << "reserved.34 " << prop.reserved[34] << std::endl;
        std::cout << "reserved.35 " << prop.reserved[35] << std::endl;
        std::cout << "reserved.36 " << prop.reserved[36] << std::endl;
        std::cout << "reserved.37 " << prop.reserved[37] << std::endl;
        std::cout << "reserved.38 " << prop.reserved[38] << std::endl;
        std::cout << "reserved.39 " << prop.reserved[39] << std::endl;
        std::cout << "reserved.40 " << prop.reserved[40] << std::endl;
        std::cout << "reserved.41 " << prop.reserved[41] << std::endl;
        std::cout << "reserved.42 " << prop.reserved[42] << std::endl;
        std::cout << "reserved.43 " << prop.reserved[43] << std::endl;
        std::cout << "reserved.44 " << prop.reserved[44] << std::endl;
        std::cout << "reserved.45 " << prop.reserved[45] << std::endl;
        std::cout << "reserved.46 " << prop.reserved[46] << std::endl;
        std::cout << "reserved.47 " << prop.reserved[47] << std::endl;
        std::cout << "reserved.48 " << prop.reserved[48] << std::endl;
        std::cout << "reserved.49 " << prop.reserved[49] << std::endl;
        std::cout << "reserved.50 " << prop.reserved[50] << std::endl;
        std::cout << "reserved.51 " << prop.reserved[51] << std::endl;
        std::cout << "reserved.52 " << prop.reserved[52] << std::endl;
        std::cout << "reserved.53 " << prop.reserved[53] << std::endl;
        std::cout << "reserved.54 " << prop.reserved[54] << std::endl;
        std::cout << "reserved.55 " << prop.reserved[55] << std::endl;
        std::cout << "reserved.56 " << prop.reserved[56] << std::endl;
        std::cout << "reserved.57 " << prop.reserved[57] << std::endl;
        std::cout << "reserved.58 " << prop.reserved[58] << std::endl;
        std::cout << "reserved.59 " << prop.reserved[59] << std::endl;
    }
    return 0;
}
